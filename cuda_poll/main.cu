#include "hip/hip_runtime.h"
#include <iostream>
#include "scheduler.hpp"

template <typename T>
__global__
void gpu_count(T* count) {
  ++(*count);
}

Task TaskA(Scheduler& sch) {

  std::cout << "Start TaskA\n";
  int* counter;
  hipStream_t stream;

  hipMallocManaged(&counter, sizeof(int));
  hipStreamCreate(&stream);
  gpu_count<<<8, 256, 0, stream>>>(counter);
 
  while(hipStreamQuery(stream) != hipSuccess) {
    co_await sch.suspend();
  }

  std::cout << "TaskA is finished\n";
  hipFreeAsync(counter, stream);
  hipStreamDestroy(stream);
}

Task TaskB(Scheduler& sch) {

  std::cout << "Start TaskB\n";
  int* counter;
  hipStream_t stream;

  hipMallocManaged(&counter, sizeof(int));
  hipStreamCreate(&stream);
  gpu_count<<<8, 256, 0, stream>>>(counter);
 
  while(hipStreamQuery(stream) != hipSuccess) {
    co_await sch.suspend();
  }

  hipFreeAsync(counter, stream);
  hipStreamDestroy(stream);

  std::cout << "TaskB is finished\n";
}


int main() {

  Scheduler sch;

  sch.emplace(TaskA(sch).get_handle());
  sch.emplace(TaskB(sch).get_handle());

  std::cout << "Start scheduling...\n";

  sch.schedule();
  sch.wait();

}


