#include "hip/hip_runtime.h"
#include <iostream>
#include "scheduler.hpp"

using namespace cudaWoCoro;

template <typename T>
__global__
void gpu_count(T* count) {
  ++(*count);
}

void TaskA() {

  std::cout << "Start TaskA\n";
  int* counter;
  hipStream_t stream;

  hipMallocManaged(&counter, sizeof(int));
  hipStreamCreate(&stream);
  gpu_count<<<8, 256, 0, stream>>>(counter);
 
  hipStreamSynchronize(stream);

  std::cout << "TaskA is finished\n";
  hipFreeAsync(counter, stream);
  hipStreamDestroy(stream);
}

void TaskB() {

  std::cout << "Start TaskB\n";
  int* counter;
  hipStream_t stream;

  hipMallocManaged(&counter, sizeof(int));
  hipStreamCreate(&stream);
  gpu_count<<<8, 256, 0, stream>>>(counter);
 
  hipStreamSynchronize(stream);

  hipFreeAsync(counter, stream);
  hipStreamDestroy(stream);

  std::cout << "TaskB is finished\n";
}


int main() {

  Scheduler sch(1);

  sch.emplace(TaskA);
  sch.emplace(TaskB);

  std::cout << "Start scheduling...\n";

  sch.schedule();
  sch.wait();

}


