#include "hip/hip_runtime.h"
#include <iostream>
#include "scheduler.hpp"

using namespace cudaCallback;

template <typename T>
__global__
void gpu_count(T* count) {
  ++(*count);
}

Task TaskA(Scheduler& sch) {
  std::cout << "Start TaskA\n";
  int* counter;
  hipStream_t stream;

  hipMallocManaged(&counter, sizeof(int));
  hipStreamCreate(&stream);
  gpu_count<<<8, 256, 0, stream>>>(counter);
 
  co_await sch.suspend(stream);

  std::cout << "TaskA is finished\n";
  hipFreeAsync(counter, stream);
  hipStreamDestroy(stream);
}

Task TaskB(Scheduler& sch) {

  std::cout << "Start TaskB\n";
  int* counter;
  hipStream_t stream;

  hipMallocManaged(&counter, sizeof(int));
  hipStreamCreate(&stream);
  gpu_count<<<8, 256, 0, stream>>>(counter);
 
  co_await sch.suspend(stream);

  hipFreeAsync(counter, stream);
  hipStreamDestroy(stream);

  std::cout << "TaskB is finished\n";
}


int main() {

  Scheduler sch;

  sch.emplace(TaskA(sch).get_handle());
  sch.emplace(TaskB(sch).get_handle());

  std::cout << "Start scheduling...\n";

  sch.schedule();
  sch.wait();

}


